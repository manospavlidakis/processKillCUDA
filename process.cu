
#include <hip/hip_runtime.h>
#include <unistd.h>     /* Symbolic Constants */
#include <iostream>
#include <sys/types.h>  /* Primitive System Data Types */
#include <errno.h>      /* Errors */
#include <stdio.h>      /* Input/Output */
#include <sys/wait.h>   /* Wait for Process Termination */
#include <stdlib.h>     /* General Utilities */
#include <semaphore.h>
#include <sys/wait.h>
#include <sys/time.h>
#include <vector>
#include <limits>
#include <iomanip>
struct timeval cuda_st, cuda_end;
using namespace std;
vector<pid_t> pidChild;
#define cudaCheckErrors(msg) \
	do { \
		hipError_t __err = hipGetLastError(); \
		if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
					msg, hipGetErrorString(__err), \
					__FILE__, __LINE__); \
			fprintf(stderr, "*** FAILED - ABORTING\n"); \
			exit(1); \
		} \
	} while (0)


__global__ void addkernel(int *data){
	for (int i=0; i<5000000000; i++)
		*data += 1;
}
void signalHandler(int signum)
{
	if (signum == SIGTERM)
	{
		std::cerr<<"Termination signal!"<<std::endl;
		exit(signum);
	}
}
void Malloc_Memcpy_Krnl()
{
	hipError_t err;
	int *h_a, *d_a;
	h_a = (int *)malloc(sizeof(int));
	err = hipMalloc(&d_a, sizeof(int));
	if (err != hipSuccess)
	{	
		cerr<<"Error malloc"<<endl;
	}
	*h_a = 1;
	err = hipMemcpy(d_a, h_a, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		cerr<<"Error memcpy"<<endl;
	}

	addkernel<<<1,1>>>(d_a);
	hipMemcpy(h_a, d_a, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_a);
}

int main()
{
	pid_t childpid; /* variable to store the child's pid */
	int status;  
	for(int i=0; i<4; i++)
	{
		childpid = fork();
		pidChild.push_back(childpid);
		if (childpid == 0) /* fork() returns 0 to the child process */
		{
			gettimeofday(&cuda_st,NULL);
			// gets time in us (convert sec -> us)
			double t1 = cuda_st.tv_sec  * 1000000 +  cuda_st.tv_usec;

			cerr << fixed << showpoint << setprecision(1)<< t1 <<" (usec) I am child wiht pid : "<<getpid()<<" assigned to device: "<<0<<endl;

			hipSetDevice(0);
			sleep(5);
			signal(SIGTERM, signalHandler);

			Malloc_Memcpy_Krnl();
		}
		else
		{
			usleep(1000);	
			cout<<"Parent: "<<getpid()<<" kill: "<<childpid<<endl;
			kill(childpid, SIGTERM);

			wait(&status);

			gettimeofday(&cuda_end,NULL);
			double t2 = cuda_end.tv_sec  * 1000000 +  cuda_end.tv_usec;
			cerr << fixed << showpoint << setprecision(1) <<"Done with kernel at "<< t2 <<" (usec)"<<endl;
		}
	}

}	

